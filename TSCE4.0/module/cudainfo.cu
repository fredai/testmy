#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <host_defines.h>
#include <string.h>
#include <stdio.h>
#include "cudainfo.h"

#if CUDA_VERSION < 5050
#error CUDA 1.x - 5.0 are not supported any more! Please use CUDA Toolkit 5.5+ instead.
#endif

#define CZ_COPY_BUF_SIZE	(16 * (1 << 20))	/*!< Transfer buffer size. */
#define CZ_COPY_LOOPS_NUM	8			/*!< Number of loops to run transfer test to. */

#define CZ_CALC_BLOCK_LOOPS	16			/*!< Number of loops to run calculation loop. */
#define CZ_CALC_BLOCK_SIZE	256			/*!< Size of instruction block. */
#define CZ_CALC_BLOCK_NUM	16			/*!< Number of instruction blocks in loop. */
#define CZ_CALC_OPS_NUM		2			/*!< Number of operations per one loop. */
#define CZ_CALC_LOOPS_NUM	8			/*!< Number of loops to run performance test to. */

#define CZ_DEF_WARP_SIZE	32			/*!< Default warp size value. */
#define CZ_DEF_THREADS_MAX	512			/*!< Default max threads value value. */

#define CZ_VER_STR_LEN		256			/*!< Version string length. */

/*!	\brief Error handling of CUDA RT calls.
*/
#define CZ_CUDA_CALL(funcCall, errProc) \
	{ \
		hipError_t errCode; \
		if((errCode = (funcCall)) != hipSuccess) { \
			printf("CUDA Error: %08x %s", errCode, hipGetErrorString(errCode)); \
			errProc; \
		} \
	}
 
/*!	\brief Check how many CUDA-devices are present.
	\return number of CUDA-devices in case of success, \a 0 if no CUDA-devies were found.
*/
int CZCudaDeviceFound(void) 
{
	int count;

	CZ_CUDA_CALL(hipGetDeviceCount(&count),
		return 0);

	return count;
}

/*!	\def ConvertSMVer2Cores(major, minor)
	\brief Get number of CUDA cores per multiprocessor.
	\arg[in] major GPU Architecture major version.
	\arg[in] minor GPU Architecture minor version.
	\returns 0 if GPU Architecture is unknown, or number of CUDA cores per multiprocessor.
*/
#define ConvertSMVer2Cores(major, minor) \
	(((major) == 1)? ( /* Tesla */ \
		((minor) == 0)? 8: /* G80*/ \
		((minor) == 1)? 8: /* G8x, G9x */ \
		((minor) == 2)? 8: /* GT21x */ \
		((minor) == 3)? 8: /* GT200 */ \
		0): \
	((major) == 2)? ( /* Fermi */ \
		((minor) == 0)? 32: /* GF100, GF110 */ \
		((minor) == 1)? 48: /* GF10x, FG11x */ \
		0): \
	((major) == 3)? ( /* Kepler */ \
		((minor) == 0)? 192: /* GK10x */ \
		((minor) == 2)? 192: /* Tegra K1 */ \
		((minor) == 5)? 192: /* GK11x, GK208 */ \
		((minor) == 7)? 192: /* GK210 */ \
		0): \
	((major) == 5)? ( /* Maxwell */ \
		((minor) == 0)? 128: /* GM10X */ \
		((minor) == 2)? 128: /* GM20X */ \
		((minor) == 3)? 128: /* Tegra X1 */ \
		0): \
	0)


void getValue1024(double value, char *valueStr, int prefix = 0, const char *unit = "B")
{
	const int prefixBase = 1024;
	int resPrefix = prefix;

	static const char *prefixTab[9] = {
		"",	/* prefixNothing */
		"K",	/* prefixK */
		"M",	/* prefixM */
		"G",	/* prefixG */
		"T",	/* prefixT */
		"P",	/* prefixP */
		"E",	/* prefixE */
		"Z",	/* prefixZ */
		"Y",	/* prefixY */
	};

	while((value > prefixBase) && (resPrefix < 9)) 
	{
		value /= prefixBase;
		resPrefix++;
	}
	sprintf(valueStr, "%.2f %s%s", value, prefixTab[resPrefix], unit);
}

/*!	\brief Local service data structure for bandwith calulations.
 * */
struct CZDeviceInfoBandLocalData {
	void		*memHostPage;	/*!< Pageable host memory. */
	void		*memHostPin;	/*!< Pinned host memory. */
	void		*memDevice1;	/*!< Device memory buffer 1. */
	void		*memDevice2;	/*!< Device memory buffer 2. */
};

/*!	\brief Set device for current thread.
 * */
int CZCudaCalcDeviceSelect(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
)
{
	CZ_CUDA_CALL(hipSetDevice(info->num),
		return -1);

	return 0;
}

/*!	\brief Allocate buffers for bandwidth calculations.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
static int CZCudaCalcDeviceBandwidthAlloc(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
)
{
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	if(info->band.localData == NULL)
    {

		//CZLog(CZLogLevelLow, "Alloc local buffers for %s.", info->deviceName);

		lData = (CZDeviceInfoBandLocalData*)malloc(sizeof(*lData));
		if(lData == NULL) 
		{
			return -1;
		}

		//CZLog(CZLogLevelLow, "Alloc host pageable for %s.", info->deviceName);

		lData->memHostPage = (void*)malloc(CZ_COPY_BUF_SIZE);
		if(lData->memHostPage == NULL) 
		{
			free(lData);
			return -1;
		}

		//CZLog(CZLogLevelLow, "Host pageable is at 0x%08X.", lData->memHostPage);

		//CZLog(CZLogLevelLow, "Alloc host pinned for %s.", info->deviceName);

		CZ_CUDA_CALL(hipHostMalloc((void**)&lData->memHostPin, CZ_COPY_BUF_SIZE),
			free(lData->memHostPage);
			free(lData);
			return -1);

		//CZLog(CZLogLevelLow, "Host pinned is at 0x%08X.", lData->memHostPin);

		//CZLog(CZLogLevelLow, "Alloc device buffer 1 for %s.", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice1, CZ_COPY_BUF_SIZE),
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		//CZLog(CZLogLevelLow, "Device buffer 1 is at 0x%08X.", lData->memDevice1);

		//CZLog(CZLogLevelLow, "Alloc device buffer 2 for %s.", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice2, CZ_COPY_BUF_SIZE),
			hipFree(lData->memDevice1);
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		//CZLog(CZLogLevelLow, "Device buffer 2 is at 0x%08X.", lData->memDevice2);

		info->band.localData = (void*)lData;
	}

	return 0;
}

/*!	\brief Free buffers for bandwidth calculations.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
static int CZCudaCalcDeviceBandwidthFree(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) 
{
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;
	if(lData != NULL) 
	{

		//CZLog(CZLogLevelLow, "Free host pageable for %s.", info->deviceName);

		if(lData->memHostPage != NULL)
			free(lData->memHostPage);

		//CZLog(CZLogLevelLow, "Free host pinned for %s.", info->deviceName);

		if(lData->memHostPin != NULL)
			hipHostFree(lData->memHostPin);

		//CZLog(CZLogLevelLow, "Free device buffer 1 for %s.", info->deviceName);

		if(lData->memDevice1 != NULL)
			hipFree(lData->memDevice1);

		//CZLog(CZLogLevelLow, "Free device buffer 2 for %s.", info->deviceName);

		if(lData->memDevice2 != NULL)
			hipFree(lData->memDevice2);

		//CZLog(CZLogLevelLow, "Free local buffers for %s.", info->deviceName);

		free(lData);
	}
	info->band.localData = NULL;

	return 0;
}

/*!	\brief Reset results of bandwidth calculations.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
static int CZCudaCalcDeviceBandwidthReset(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->band.copyHDPage = 0;
	info->band.copyHDPin = 0;
	info->band.copyDHPage = 0;
	info->band.copyDHPin = 0;
	info->band.copyDD = 0;

	return 0;
}

#define CZ_COPY_MODE_H2D	0	/*!< Host to device data copy mode. */
#define CZ_COPY_MODE_D2H	1	/*!< Device to host data copy mode. */
#define CZ_COPY_MODE_D2D	2	/*!< Device to device data copy mode. */

/*!	\brief Run data transfer bandwidth tests.
 * 	\return \a 0 in case of success, \a other is value in KiB/s.
 * 	*/
static float CZCudaCalcDeviceBandwidthTestCommon (
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int mode,			/*!<[in] Run bandwidth test in one of modes. */
	int pinned			/*!<[in] Use pinned \a (=1) memory buffer instead of pagable \a (=0). */
) 
{
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKiBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice1;
	void *memDevice2;
	int i;

	if(info == NULL)
		return 0;

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? lData->memHostPin: lData->memHostPage;
	memDevice1 = lData->memDevice1;
	memDevice2 = lData->memDevice2;

	/*CZLog(CZLogLevelLow, "Starting %s test (%s) on %s.",
		(mode == CZ_COPY_MODE_H2D)? "host to device":
		(mode == CZ_COPY_MODE_D2H)? "device to host":
		(mode == CZ_COPY_MODE_D2D)? "device to device": "unknown",
		pinned? "pinned": "pageable",
		info->deviceName);*/

	for(i = 0; i < CZ_COPY_LOOPS_NUM; i++) {

		float loopMs = 0.0;

		CZ_CUDA_CALL(hipEventRecord(start, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		switch(mode) {
		case CZ_COPY_MODE_H2D:
			CZ_CUDA_CALL(hipMemcpy(memDevice1, memHost, CZ_COPY_BUF_SIZE, hipMemcpyHostToDevice),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		case CZ_COPY_MODE_D2H:
			CZ_CUDA_CALL(hipMemcpy(memHost, memDevice2, CZ_COPY_BUF_SIZE, hipMemcpyDeviceToHost),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		case CZ_COPY_MODE_D2D:
			CZ_CUDA_CALL(hipMemcpy(memDevice2, memDevice1, CZ_COPY_BUF_SIZE, hipMemcpyDeviceToDevice),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		default: // WTF!
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}

		CZ_CUDA_CALL(hipEventRecord(stop, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventSynchronize(stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventElapsedTime(&loopMs, start, stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		timeMs += loopMs;
	}

	//CZLog(CZLogLevelLow, "Test complete in %f ms.", timeMs);

	bandwidthKiBs = (
		1000 *
		(float)CZ_COPY_BUF_SIZE *
		(float)CZ_COPY_LOOPS_NUM
	) / (
		timeMs *
		(float)(1 << 10)
	);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return bandwidthKiBs;
}

/*!	\brief Run several bandwidth tests.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
static int CZCudaCalcDeviceBandwidthTest(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) 
{

	info->band.copyHDPage = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_H2D, 0);
	info->band.copyHDPin = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_H2D, 1);
	info->band.copyDHPage = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2H, 0);
	info->band.copyDHPin = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2H, 1);
	info->band.copyDD = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2D, 0);

	return 0;
}

/*!	\brief Calculate bandwidth information about CUDA-device.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
int CZCudaCalcDeviceBandwidth(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;
		
	if (CZCudaCalcDeviceSelect(info) != 0)
		return -1;

	if(CZCudaCalcDeviceBandwidthReset(info) != 0)
		return -1;

	if(CZCudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	if(CZCudaCalcDeviceBandwidthTest(info) != 0)
		return -1;

	// CZCudaCalcDeviceBandwidthFree(info);
	return 0;
}

/*!	\brief Cleanup after test and bandwidth calculations.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
int CZCudaCleanDevice(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) 
{

	if(info == NULL)
		return -1;

	if(CZCudaCalcDeviceBandwidthFree(info) != 0)
		return -1;

	return 0;
}

/*!	\brief Reset results of preformance calculations.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
static int CZCudaCalcDevicePerformanceReset(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) 
{

	if(info == NULL)
		return -1;

	info->perf.calcFloat = 0;
	info->perf.calcDouble = 0;
	info->perf.calcInteger32 = 0;
	info->perf.calcInteger24 = 0;
	info->perf.calcInteger64 = 0;

	return 0;
}

/*!	\brief 16 MAD instructions for float point test.
 * */
#define CZ_CALC_FMAD_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

/*!	\brief 256 MAD instructions for float point test.
 * */
#define CZ_CALC_FMAD_256(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \

/*!	\brief 16 DMAD instructions for double-precision test.
 * */
#define CZ_CALC_DFMAD_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

/*	a = fma(a, a, a); b = fma(b, b, b); a = fma(a, a, a); b = fma(b, b, b); \
 *		a = fma(a, a, a); b = fma(b, b, b); a = fma(a, a, a); b = fma(b, b, b); \
 *			a = fma(a, a, a); b = fma(b, b, b); a = fma(a, a, a); b = fma(b, b, b); \
 *				a = fma(a, a, a); b = fma(b, b, b); a = fma(a, a, a); b = fma(b, b, b); \*/

/*!	\brief 256 MAD instructions for float point test.
 * */
#define CZ_CALC_DFMAD_256(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \

/*!	\brief 16 MAD instructions for 32-bit integer test.
 * */
#define CZ_CALC_IMAD32_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

/*!	\brief 256 MAD instructions for 32-bit integer test.
 * */
#define CZ_CALC_IMAD32_256(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \

/*!	\brief 16 MAD instructions for 64-bit integer test.
 * */
#define CZ_CALC_IMAD64_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

/*!	\brief 256 MAD instructions for 64-bit integer test.
 * */
#define CZ_CALC_IMAD64_256(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \

/*!	\brief 16 MAD instructions for 24-bit integer test.
 * */
#define CZ_CALC_IMAD24_16(a, b) \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \

/*!	\brief 256 MAD instructions for 24-bit integer test.
 * */
#define CZ_CALC_IMAD24_256(a, b) \
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\

/*!	\brief 16 MAD instructions for 8-bit integer test.
 * */
#define CZ_CALC_IMAD8_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

/*!	\brief 256 MAD instructions for 8-bit integer test.
 * */
#define CZ_CALC_IMAD8_256(a, b) \
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\
	CZ_CALC_IMAD8_16(a, b) CZ_CALC_IMAD8_16(a, b)\

#define CZ_CALC_MODE_FLOAT	0	/*!< Single-precision float point test mode. */
#define CZ_CALC_MODE_DOUBLE	1	/*!< Double-precision float point test mode. */
#define CZ_CALC_MODE_INTEGER8	2	/*!< 8-bit integer test mode. */
#define CZ_CALC_MODE_INTEGER32	3	/*!< 32-bit integer test mode. */
#define CZ_CALC_MODE_INTEGER24	4	/*!< 24-bit integer test mode. */
#define CZ_CALC_MODE_INTEGER64	5	/*!< 64-bit integer test mode. */

/*!	\brief GPU code for float point test.
 * */
 __global__ void CZCudaCalcKernelFloat(
	void *buf			/*!<[in] Data buffer. */
) 
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float *arr = (float*)buf;
	float val1 = index;
	float val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for double-precision test.
 * */
__global__ void CZCudaCalcKernelDouble(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	double *arr = (double*)buf;
	double val1 = index;
	double val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for 8-bit integer test.
 * */
__global__ void CZCudaCalcKernelInteger8(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	char *arr = (char*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
		CZ_CALC_IMAD8_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for 32-bit integer test.
 * */
__global__ void CZCudaCalcKernelInteger32(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for 24-bit integer test.
 * */
__global__ void CZCudaCalcKernelInteger24(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief GPU code for 64-bit integer test.
 * */
__global__ void CZCudaCalcKernelInteger64(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	long long *arr = (long long*)buf;
	long long val1 = index;
	long long val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

/*!	\brief Run GPU calculation performace tests.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
static float CZCudaCalcDevicePerformanceTest(
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int mode			/*!<[in] Run performance test in one of modes. */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float performanceKOPs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	int blocksNum = info->heavyMode? info->core.muliProcCount: 1;
	int i;

	if(info == NULL)
		return 0;

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	if (info->band.localData == NULL)
	{
		lData = (CZDeviceInfoBandLocalData*)malloc(sizeof(CZDeviceInfoBandLocalData));
		if(lData == NULL) 
		{
			return 0;
		}
		memset(lData, 0, sizeof(CZDeviceInfoBandLocalData));
		info->band.localData = lData;
	}
	if (lData->memDevice1 == NULL)
	{
		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice1, CZ_COPY_BUF_SIZE),
			free(lData);
			return 0);
	}

	int threadsNum = info->core.maxThreadsPerBlock;
	if(threadsNum == 0) {
		int warpSize = info->core.SIMDWidth;
		if(warpSize == 0)
			warpSize = CZ_DEF_WARP_SIZE;
		threadsNum = warpSize * 2;
		if(threadsNum > CZ_DEF_THREADS_MAX)
			threadsNum = CZ_DEF_THREADS_MAX;
	}

/*	CZLog(CZLogLevelLow, "Starting %s test on %s on %d block(s) %d thread(s) each.",
		(mode == CZ_CALC_MODE_FLOAT)? "single-precision float":
		(mode == CZ_CALC_MODE_DOUBLE)? "double-precision float":
		(mode == CZ_CALC_MODE_INTEGER8)? "8-bit integer":
		(mode == CZ_CALC_MODE_INTEGER32)? "32-bit integer":
		(mode == CZ_CALC_MODE_INTEGER24)? "24-bit integer":
		(mode == CZ_CALC_MODE_INTEGER64)? "64-bit integer": "unknown",
		info->deviceName,
		blocksNum,
		threadsNum);*/

	for(i = 0; i < CZ_CALC_LOOPS_NUM; i++) {

		float loopMs = 0.0;

		CZ_CUDA_CALL(hipEventRecord(start, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		switch(mode) {
		case CZ_CALC_MODE_FLOAT:
			CZCudaCalcKernelFloat<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_DOUBLE:
			CZCudaCalcKernelDouble<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER8:
			CZCudaCalcKernelInteger8<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER32:
			CZCudaCalcKernelInteger32<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER24:
			CZCudaCalcKernelInteger24<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER64:
			CZCudaCalcKernelInteger64<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		default: // WTF!
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}

		CZ_CUDA_CALL(hipGetLastError(),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventRecord(stop, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventSynchronize(stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventElapsedTime(&loopMs, start, stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		timeMs += loopMs;
	}

	//CZLog(CZLogLevelLow, "Test complete in %f ms.", timeMs);

	performanceKOPs = (
		(float)info->core.muliProcCount *
		(float)CZ_CALC_LOOPS_NUM *
		(float)threadsNum *
		(float)CZ_CALC_BLOCK_LOOPS *
		(float)CZ_CALC_OPS_NUM *
		(float)CZ_CALC_BLOCK_SIZE *
		(float)CZ_CALC_BLOCK_NUM
	) / (float)timeMs;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	CZCudaCalcDeviceBandwidthFree(info);
	return performanceKOPs;
}

/*!	\brief Calculate performance information about CUDA-device.
 * 	\return \a 0 in case of success, \a -1 in case of error.
 * 	*/
int CZCudaCalcDevicePerformance(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;
		
	if (CZCudaCalcDeviceSelect(info) != 0)
		return -1;
	
	if(CZCudaCalcDevicePerformanceReset(info) != 0)
		return -1;

	info->perf.calcFloat = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_FLOAT);
	if(((info->major > 1)) ||
		((info->major == 1) && (info->minor >= 3)))
		info->perf.calcDouble = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_DOUBLE);
	//info->perf.calcInteger8 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER8);
	info->perf.calcInteger32 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER32);
	info->perf.calcInteger24 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER24);
	info->perf.calcInteger64 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER64);

	return 0;
}

/*!	\brief Read information about a CUDA-device.
	\return \a 0 in case of success, \a -1 in case of error.
*/
int CZCudaReadDeviceInfo(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
)
{
	hipDeviceProp_t prop;

	if(info == NULL)
		return -1;

	if(info->num >= CZCudaDeviceFound())
		return -1;

	CZ_CUDA_CALL(hipGetDeviceProperties(&prop, info->num),
		return -1);

	strcpy(info->deviceName, prop.name);
	info->major = prop.major;
	info->minor = prop.minor;

	info->core.regsPerBlock = prop.regsPerBlock;
	info->core.regsPerMultipro = prop.regsPerMultiprocessor;
	info->core.SIMDWidth = prop.warpSize;
	info->core.maxThreadsPerBlock = prop.maxThreadsPerBlock;
	info->core.maxThreadsDim[0] = prop.maxThreadsDim[0];
	info->core.maxThreadsDim[1] = prop.maxThreadsDim[1];
	info->core.maxThreadsDim[2] = prop.maxThreadsDim[2];
	info->core.maxGridSize[0] = prop.maxGridSize[0];
	info->core.maxGridSize[1] = prop.maxGridSize[1];
	info->core.maxGridSize[2] = prop.maxGridSize[2];
	info->core.clockRate = prop.clockRate/1000;
	info->core.muliProcCount = prop.multiProcessorCount;
	info->core.kernelExecTimeoutEnabled= prop.kernelExecTimeoutEnabled;
	info->core.integratedGpu = prop.integrated;
	info->core.concurrentKernels = prop.concurrentKernels;
	info->core.computeMode =
		(prop.computeMode == hipComputeModeDefault)? CZComputeModeDefault:
		(prop.computeMode == hipComputeModeExclusive)? CZComputeModeExclusive:
		(prop.computeMode == hipComputeModeProhibited)? CZComputeModeProhibited:
		CZComputeModeUnknown;
	info->core.pciBusID = prop.pciBusID;
	info->core.pciDeviceID = prop.pciDeviceID;
	info->core.pciDomainID = prop.pciDomainID;
	info->core.maxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
	info->core.cudaCores = ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount;
	info->core.streamPrioritiesSupported = prop.streamPrioritiesSupported;

	info->mem.totalGlobal = prop.totalGlobalMem;
	info->mem.sharedPerBlock = prop.sharedMemPerBlock;
	info->mem.sharedPerMultiProcessor = prop.sharedMemPerMultiprocessor;
	info->mem.maxPitch = prop.memPitch;
	info->mem.totalConst = prop.totalConstMem;
	info->mem.textureAlignment = prop.textureAlignment;
	info->mem.texture1D[0] = prop.maxTexture1D;
	info->mem.texture2D[0] = prop.maxTexture2D[0];
	info->mem.texture2D[1] = prop.maxTexture2D[1];
	info->mem.texture3D[0] = prop.maxTexture3D[0];
	info->mem.texture3D[1] = prop.maxTexture3D[1];
	info->mem.texture3D[2] = prop.maxTexture3D[2];
	info->mem.gpuOverlap = prop.deviceOverlap;
	info->mem.mapHostMemory = prop.canMapHostMemory;
	info->mem.errorCorrection = prop.ECCEnabled;
	info->mem.asyncEngineCount = prop.asyncEngineCount;
	info->mem.unifiedAddressing = prop.unifiedAddressing;
	info->mem.memoryClockRate = prop.memoryClockRate/1000;
	info->mem.memoryBusWidth = prop.memoryBusWidth;
	info->mem.l2CacheSize = prop.l2CacheSize;

	return 0;
}

void printInfo(CZDeviceInfo *info)
{
	char valueStr[30];
	char valueStrPf[30];
	printf("=====================================\n");
	printf("DeviceName[%d]:%s\n", info->num, info->deviceName);
	printf("************Core Info*************\n");
	printf("Compute Capability:%d.%d\n", info->major, info->minor);
	printf("Clock Rate:%d MHz\n", info->core.clockRate);
	printf("Multiprocessors:%d (%d Cores)\n", info->core.muliProcCount, info->core.cudaCores);
	printf("Cores Per Multiprocessor:%d\n", info->core.cudaCores/info->core.muliProcCount);
	printf("WarpSize:%d\n", info->core.SIMDWidth);
	printf("Max Threads Per Multiprocessor:%d\n", info->core.maxThreadsPerMultiProcessor);
	printf("Max Threads Per Block:%d\n", info->core.maxThreadsPerBlock);
	printf("Regs Per Block:%d\n", info->core.regsPerBlock);
	printf("maxThreadsDim:%dx%dx%d\n", info->core.maxThreadsDim[0], info->core.maxThreadsDim[1], info->core.maxThreadsDim[2]);
	printf("maxGridSize:%dx%dx%d\n", info->core.maxGridSize[0], info->core.maxGridSize[1], info->core.maxGridSize[2]);
	printf("computeMode:%d\n", info->core.computeMode);
	printf("kernelExecTimeoutEnabled:%d\n", info->core.kernelExecTimeoutEnabled);
	printf("integratedGpu:%d\n", info->core.integratedGpu);
	printf("concurrentKernels:%d\n", info->core.concurrentKernels);
	printf("streamPrioritiesSupported:%d\n", info->core.streamPrioritiesSupported);
	printf("pciBusID:%d\n", info->core.pciBusID);
	printf("pciDeviceID:%d\n", info->core.pciDeviceID);
	printf("pciDomainID:%d\n", info->core.pciDomainID);
	printf("************Memory Info*************\n");
	getValue1024(info->mem.totalGlobal, valueStr);
	printf("totalGlobalMem:%s\n", valueStr);
	getValue1024(info->mem.totalConst, valueStr);
	printf("totalConstMem:%s\n", valueStr);
	getValue1024(info->mem.sharedPerBlock, valueStr);
	printf("sharedMemPerBlock:%s \n", valueStr);
	getValue1024(info->mem.sharedPerMultiProcessor, valueStr);
	printf("sharedMemPerMultiProcessor:%s\n", valueStr);
	getValue1024(info->mem.l2CacheSize, valueStr);
	printf("l2CacheSize:%s\n", valueStr);
	printf("memoryClockRate:%d MHz\n", info->mem.memoryClockRate);
	printf("memoryBusWidth:%d bits\n", info->mem.memoryBusWidth);
	getValue1024(info->mem.maxPitch, valueStr);
	printf("maxPitch:%s \n", valueStr);
	printf("textureAlignment:%d\n", info->mem.textureAlignment);
	printf("texture1D Size:%d\n", info->mem.texture1D[0]);
	printf("texture2D Size:%dx%d\n", info->mem.texture2D[0], info->mem.texture2D[1]);
	printf("texture3D Size:%dx%dx%d\n", info->mem.texture3D[0], info->mem.texture3D[1], info->mem.texture3D[2]);
	printf("errorCorrection:%d\n", info->mem.errorCorrection);
	printf("mapHostMemory:%d\n", info->mem.mapHostMemory);
	printf("unifiedAddressing:%d\n", info->mem.unifiedAddressing);
	printf("gpuOverlap:%d\n", info->mem.gpuOverlap);
	printf("asyncEngineCount:%d\n", info->mem.asyncEngineCount);
	printf("************Performace Info*************\n");
	printf("MemoryCopy		Pinned		Pageable\n");
	getValue1024(info->band.copyHDPin, valueStr, 1, "B/s");
	getValue1024(info->band.copyHDPage, valueStrPf, 1, "B/s");
	printf("HostToDevice	%s			%s\n", valueStr, valueStrPf);
	getValue1024(info->band.copyDHPin, valueStr, 1, "B/s");;
	getValue1024(info->band.copyDHPage, valueStrPf, 1, "B/s");
	printf("DeviceToHost	%s			%s\n", valueStr, valueStrPf);
	getValue1024(info->band.copyDD, valueStr, 1, "B/s");
	printf("DeviceToDevice	%s\n", valueStr);
	printf("GPU Core Performace\n");
	getValue1024(info->perf.calcFloat, valueStr, 1, "flop/s");
	printf("Single-precision Float	%s\n", valueStr);
	getValue1024(info->perf.calcDouble, valueStr, 1, "flop/s");
	printf("Double-precision Float	%s\n", valueStr);
	getValue1024(info->perf.calcInteger64, valueStr, 1, "iop/s");
	printf("64-bit Integer			%s\n", valueStr);
	getValue1024(info->perf.calcInteger32, valueStr, 1, "iop/s");
	printf("32-bit Integer			%s\n", valueStr);
	getValue1024(info->perf.calcInteger24, valueStr, 1, "iop/s");
	printf("24-bit Integer			%s\n", valueStr);
}
